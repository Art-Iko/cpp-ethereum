#include "hip/hip_runtime.h"
/*
* Genoil's CUDA mining kernel for Ethereum
* based on Tim Hughes' opencl kernel.
* thanks to trpuvot,djm34,sp,cbuchner for things i took from ccminer. 
*/

#include "ethash_cu_miner_kernel.h"
#include "ethash_cu_miner_kernel_globals.h"
#include "rotl64.cuh"
#include ""
#include "hip/device_functions.h"
#include "hip/hip_vector_types.h"

#define GROUP_SIZE 512
#define ACCESSES 64
#define THREADS_PER_HASH (128 / 16)
#define HASHES_PER_LOOP (GROUP_SIZE / THREADS_PER_HASH)

#define FNV_PRIME	0x01000193

__device__ __constant__ ulong const keccak_round_constants[24] = {
	0x0000000000000001ULL, 0x0000000000008082ULL, 0x800000000000808AULL,
	0x8000000080008000ULL, 0x000000000000808BULL, 0x0000000080000001ULL,
	0x8000000080008081ULL, 0x8000000000008009ULL, 0x000000000000008AULL,
	0x0000000000000088ULL, 0x0000000080008009ULL, 0x000000008000000AULL,
	0x000000008000808BULL, 0x800000000000008BULL, 0x8000000000008089ULL,
	0x8000000000008003ULL, 0x8000000000008002ULL, 0x8000000000000080ULL,
	0x000000000000800AULL, 0x800000008000000AULL, 0x8000000080008081ULL,
	0x8000000000008080ULL, 0x0000000080000001ULL, 0x8000000080008008ULL
};

__device__ static void keccak_f1600_block(ulong* s, uint out_size)//, uint in_size, uint out_size)
{
	ulong t[5], u, v;
	
	for (size_t i = 0; i < 24; i++) {
		/* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
		t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
		t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
		t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
		t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
		t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		/* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
		u = t[4] ^ ROTL64L(t[1], 1);
		s[0] ^= u; s[5] ^= u; s[10] ^= u; s[15] ^= u; s[20] ^= u;
		u = t[0] ^ ROTL64L(t[2], 1);
		s[1] ^= u; s[6] ^= u; s[11] ^= u; s[16] ^= u; s[21] ^= u;
		u = t[1] ^ ROTL64L(t[3], 1);
		s[2] ^= u; s[7] ^= u; s[12] ^= u; s[17] ^= u; s[22] ^= u;
		u = t[2] ^ ROTL64L(t[4], 1);
		s[3] ^= u; s[8] ^= u; s[13] ^= u; s[18] ^= u; s[23] ^= u;
		u = t[3] ^ ROTL64L(t[0], 1);
		s[4] ^= u; s[9] ^= u; s[14] ^= u; s[19] ^= u; s[24] ^= u;
		 
		/* rho pi: b[..] = rotl(a[..], ..) */
		u = s[1];

		s[1] = ROTL64H(s[6], 44);
		s[6] = ROTL64L(s[9], 20);
		s[9] = ROTL64H(s[22], 61);
		s[22] = ROTL64H(s[14], 39);
		s[14] = ROTL64L(s[20], 18);
		s[20] = ROTL64H(s[2], 62);
		s[2] = ROTL64H(s[12], 43);
		s[12] = ROTL64L(s[13], 25);
		s[13] = ROTL64L(s[19], 8);
		s[19] = ROTL64H(s[23], 56);
		s[23] = ROTL64H(s[15], 41);
		s[15] = ROTL64L(s[4], 27);
		s[4] = ROTL64L(s[24], 14);
		s[24] = ROTL64L(s[21], 2);
		s[21] = ROTL64H(s[8], 55);
		s[8] = ROTL64H(s[16], 45);
		s[16] = ROTL64H(s[5], 36);
		s[5] = ROTL64L(s[3], 28);
		s[3] = ROTL64L(s[18], 21);
		s[18] = ROTL64L(s[17], 15);
		s[17] = ROTL64L(s[11], 10);
		s[11] = ROTL64L(s[7], 6);
		s[7] = ROTL64L(s[10], 3);
		s[10] = ROTL64L(u, 1);

		/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
		u = s[0]; v = s[1]; s[0] ^= (~v) & s[2]; 
		
		// squeeze this in here
		/* iota: a[0,0] ^= round constant */
		s[0] ^= keccak_round_constants[i];

		// continue chi
		s[1] ^= (~s[2]) & s[3]; s[2] ^= (~s[3]) & s[4]; s[3] ^= (~s[4]) & u; s[4] ^= (~u) & v;
		if (i == 23 && out_size == 4) return;
		u = s[5]; v = s[6]; s[5] ^= (~v) & s[7]; s[6] ^= (~s[7]) & s[8]; s[7] ^= (~s[8]) & s[9]; 
		if (i == 23 && out_size == 8) return;
		s[8] ^= (~s[9]) & u; s[9] ^= (~u) & v;
		u = s[10]; v = s[11]; s[10] ^= (~v) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & u; s[14] ^= (~u) & v;
		u = s[15]; v = s[16]; s[15] ^= (~v) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & u; s[19] ^= (~u) & v;
		u = s[20]; v = s[21]; s[20] ^= (~v) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & u; s[24] ^= (~u) & v;
	}
}

#define copy(dst, src, count) for (uint i = 0; i < count; i++) { (dst)[i] = (src)[i]; }

#define countof(x) (sizeof(x) / sizeof(x[0]))

#define fnv(x,y) ((x) * FNV_PRIME ^(y))

__device__ uint4 fnv4(uint4 a, uint4 b)
{
	uint4 c;
	c.x = a.x * FNV_PRIME ^ b.x;	
	c.y = a.y * FNV_PRIME ^ b.y;
	c.z = a.z * FNV_PRIME ^ b.z;
	c.w = a.w * FNV_PRIME ^ b.w;
	return c;
}

__device__ uint fnv_reduce(uint4 v)
{
	return fnv(fnv(fnv(v.x, v.y), v.z), v.w);
}

__device__ hash64_t init_hash(hash32_t const* header, ulong nonce)
{
	hash64_t init;
	//uint const init_size = countof(init.ulongs);	//	8
	//uint const hash_size = countof(header->ulongs);	//	4

	// sha3_512(header .. nonce)
	ulong state[25];

	copy(state, header->ulongs, 4);
	state[4] = nonce;
	state[5] = 0x0000000000000001;
	state[6] = 0;
	state[7] = 0;
	state[8] = 0x8000000000000000;
	for (uint i = 9; i < 25; i++)
	{
		state[i] = 0;
	}
	
	keccak_f1600_block(state, 8);// , hash_size + 1, init_size);
	 
	copy(init.ulongs, state, 8);
	return init;
}

__device__ uint inner_loop(uint4 mix, uint thread_id, uint* share, hash128_t const* g_dag)
{
	// share init0
	if (thread_id == 0)
		*share = mix.x;
	//__syncthreads();
	uint init0 = *share;
	
	uint a = 0;

	do
	{
		
		bool update_share = thread_id == ((a >> 2) & (THREADS_PER_HASH-1));

		//#pragma unroll 4
		for (uint i = 0; i < 4; i++)
		{
			
			if (update_share)
			{
				uint m[4] = { mix.x, mix.y, mix.z, mix.w };
				*share = fnv(init0 ^ (a + i), m[i]) % d_dag_size;
			}
			//__syncthreads();
			__threadfence_block();

			mix = fnv4(mix, g_dag[*share].uint4s[thread_id]);
		}
		
	} while ((a += 4) != ACCESSES);// d_acceses);
	
	return fnv_reduce(mix);
}

__device__ hash32_t final_hash(hash64_t const* init, hash32_t const* mix)
{
	ulong state[25];

	hash32_t hash;
	//uint const hash_size = countof(hash.ulongs);	//	4
	//uint const init_size = countof(init->ulongs);	//	8
	//uint const mix_size  = countof(mix->ulongs);	//	4

	// keccak_256(keccak_512(header..nonce) .. mix);
	copy(state, init->ulongs, 8);
	copy(state + 8, mix->ulongs, 4);
	state[12] = 0x0000000000000001;
	for (uint i = 13; i < 16; i++)
	{
		state[i] = 0;
	}
	state[16] = 0x8000000000000000;
	for (uint i = 17; i < 25; i++)
	{
		state[i] = 0;
	}

	keccak_f1600_block(state,4);// , init_size + mix_size, hash_size);

	// copy out
	copy(hash.ulongs, state, 4);
	return hash;
}

typedef union
{
	hash64_t init;
	hash32_t mix;
} compute_hash_share;

__device__ hash32_t compute_hash(
	compute_hash_share* share,
	hash32_t const* g_header,
	hash128_t const* g_dag,
	ulong start_nonce,
	ulong gid
	)
{
	// Compute one init hash per work item.
	hash64_t init = init_hash(g_header, start_nonce + gid);

	// Threads work together in this phase in groups of 8.
	uint const thread_id = gid & (THREADS_PER_HASH-1);
	uint const hash_id = (gid & (GROUP_SIZE - 1)) >> 3;/// THREADS_PER_HASH;

	hash32_t mix;
	uint i = 0;
	
	do
	{
		// share init with other threads
		if (i == thread_id)
			share[hash_id].init = init;

		uint4 thread_init = share[hash_id].init.uint4s[(thread_id & 3)];

		uint thread_mix = inner_loop(thread_init, thread_id, share[hash_id].mix.uints, g_dag);

		share[hash_id].mix.uints[thread_id] = thread_mix;

		if (i == thread_id)
			mix = share[hash_id].mix;

	} while (++i != THREADS_PER_HASH );

	return final_hash(&init, &mix);

}

__global__ void ethash_search(
	uint* g_output,
	hash32_t const* g_header,
	hash128_t const* g_dag,
	ulong start_nonce,
	ulong target
	)
{
	__shared__ compute_hash_share share[GROUP_SIZE / THREADS_PER_HASH];
	
	uint const gid = blockIdx.x * blockDim.x + threadIdx.x;
	
	hash32_t hash = compute_hash(share, g_header, g_dag, start_nonce, gid);
	
	if (__brevll(hash.ulongs[0]) < target)
	{
		atomicInc(g_output,d_max_outputs);
		g_output[g_output[0]] = gid;
	}
	
}

void run_ethash_hash(
	hash32_t* g_hashes,
	hash32_t const* g_header,
	hash128_t const* g_dag,
	ulong start_nonce
) 
{
}

void run_ethash_search(
	uint blocks,
	uint threads,
	hipStream_t stream,
	uint* g_output,
	hash32_t const* g_header,
	hash128_t const* g_dag,
	ulong start_nonce,
	ulong target
)
{
	ethash_search<<<blocks, threads, 0, stream>>>(g_output, g_header, g_dag, start_nonce, target);
}

hipError_t set_constants(
	uint * dag_size,
	uint * max_outputs
	)
{
	hipError_t result;
	result = hipMemcpyToSymbol(HIP_SYMBOL(d_dag_size), dag_size, sizeof(uint));
	result = hipMemcpyToSymbol(HIP_SYMBOL(d_max_outputs), max_outputs, sizeof(uint));
	return result;
}
