#include "hip/hip_runtime.h"
/*
* Genoil's CUDA mining kernel for Ethereum
* based on Tim Hughes' opencl kernel.
* thanks to trpuvot,djm34,sp,cbuchner for things i took from ccminer.
*/

#include "ethash_cu_miner_kernel.h"
#include "ethash_cu_miner_kernel_globals.h"
#include "rotl64.cuh"
#include ""
#include "hip/device_functions.h"
#include "hip/hip_vector_types.h"

#define ACCESSES 64
#define THREADS_PER_HASH (128 / 16)

#define FNV_PRIME	0x01000193

// Thanks for Lukas' code here
/*
#define SWAP64(n)					\
  (((n) << 56)						\
   | (((n) & 0xff00) << 40)			\
   | (((n) & 0xff0000) << 24)		\
   | (((n) & 0xff000000) << 8)		\
   | (((n) >> 8) & 0xff000000)		\
   | (((n) >> 24) & 0xff0000)		\
   | (((n) >> 40) & 0xff00)			\
   | ((n)  >> 56))
*/

#define SWAP64(v) \
  ((ROTL64L(v,  8) & 0x000000FF000000FF) | \
   (ROTL64L(v, 24) & 0x0000FF000000FF00) | \
   (ROTL64H(v, 40) & 0x00FF000000FF0000) | \
   (ROTL64H(v, 56) & 0xFF000000FF000000))

#define PACK64(result, lo, hi) asm("mov.b64 %0, {%1,%2};//pack64"  : "=l"(result) : "r"(lo), "r"(hi));
#define UNPACK64(lo, hi, input) asm("mov.b64 {%0, %1}, %2;//unpack64" : "=r"(lo),"=r"(hi) : "l"(input));


__device__ __constant__ uint64_t const keccak_round_constants[24] = {
	0x0000000000000001ULL, 0x0000000000008082ULL, 0x800000000000808AULL,
	0x8000000080008000ULL, 0x000000000000808BULL, 0x0000000080000001ULL,
	0x8000000080008081ULL, 0x8000000000008009ULL, 0x000000000000008AULL,
	0x0000000000000088ULL, 0x0000000080008009ULL, 0x000000008000000AULL,
	0x000000008000808BULL, 0x800000000000008BULL, 0x8000000000008089ULL,
	0x8000000000008003ULL, 0x8000000000008002ULL, 0x8000000000000080ULL,
	0x000000000000800AULL, 0x800000008000000AULL, 0x8000000080008081ULL,
	0x8000000000008080ULL, 0x0000000080000001ULL, 0x8000000080008008ULL
};

__device__ static void keccak_f1600_block(uint64_t* s, uint32_t out_size)//, uint32_t in_size, uint32_t out_size)
{
	uint64_t t[5], u, v;

	for (size_t i = 0; i < 24; i++) {
		/* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
		t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
		t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
		t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
		t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
		t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		/* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
		u = t[4] ^ ROTL64L(t[1], 1);
		s[0] ^= u; s[5] ^= u; s[10] ^= u; s[15] ^= u; s[20] ^= u;
		u = t[0] ^ ROTL64L(t[2], 1);
		s[1] ^= u; s[6] ^= u; s[11] ^= u; s[16] ^= u; s[21] ^= u;
		u = t[1] ^ ROTL64L(t[3], 1);
		s[2] ^= u; s[7] ^= u; s[12] ^= u; s[17] ^= u; s[22] ^= u;
		u = t[2] ^ ROTL64L(t[4], 1);
		s[3] ^= u; s[8] ^= u; s[13] ^= u; s[18] ^= u; s[23] ^= u;
		u = t[3] ^ ROTL64L(t[0], 1);
		s[4] ^= u; s[9] ^= u; s[14] ^= u; s[19] ^= u; s[24] ^= u;
		 
		/* rho pi: b[..] = rotl(a[..], ..) */
		u = s[1];

		s[1] = ROTL64H(s[6], 44);
		s[6] = ROTL64L(s[9], 20);
		s[9] = ROTL64H(s[22], 61);
		s[22] = ROTL64H(s[14], 39);
		s[14] = ROTL64L(s[20], 18);
		s[20] = ROTL64H(s[2], 62);
		s[2] = ROTL64H(s[12], 43);
		s[12] = ROTL64L(s[13], 25);
		s[13] = ROTL64L(s[19], 8);
		s[19] = ROTL64H(s[23], 56);
		s[23] = ROTL64H(s[15], 41);
		s[15] = ROTL64L(s[4], 27);
		s[4] = ROTL64L(s[24], 14);
		s[24] = ROTL64L(s[21], 2);
		s[21] = ROTL64H(s[8], 55);
		s[8] = ROTL64H(s[16], 45);
		s[16] = ROTL64H(s[5], 36);
		s[5] = ROTL64L(s[3], 28);
		s[3] = ROTL64L(s[18], 21);
		s[18] = ROTL64L(s[17], 15);
		s[17] = ROTL64L(s[11], 10);
		s[11] = ROTL64L(s[7], 6);
		s[7] = ROTL64L(s[10], 3);
		s[10] = ROTL64L(u, 1);

		/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
		u = s[0]; v = s[1]; s[0] ^= (~v) & s[2]; 
		
		// squeeze this in here
		/* iota: a[0,0] ^= round constant */
		s[0] ^= keccak_round_constants[i];

		// continue chi
		s[1] ^= (~s[2]) & s[3]; s[2] ^= (~s[3]) & s[4]; s[3] ^= (~s[4]) & u; s[4] ^= (~u) & v;
		if (i == 23 && out_size == 4) return;
		u = s[5]; v = s[6]; s[5] ^= (~v) & s[7]; s[6] ^= (~s[7]) & s[8]; s[7] ^= (~s[8]) & s[9]; 
		if (i == 23 && out_size == 8) return;
		s[8] ^= (~s[9]) & u; s[9] ^= (~u) & v;
		u = s[10]; v = s[11]; s[10] ^= (~v) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & u; s[14] ^= (~u) & v;
		u = s[15]; v = s[16]; s[15] ^= (~v) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & u; s[19] ^= (~u) & v;
		u = s[20]; v = s[21]; s[20] ^= (~v) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & u; s[24] ^= (~u) & v;
	}
}

#define copy(dst, src, count) for (uint32_t i = 0; i < count; i++) { (dst)[i] = (src)[i]; }

#define countof(x) (sizeof(x) / sizeof(x[0]))

#define fnv(x,y) ((x) * FNV_PRIME ^(y))

__device__ uint4 fnv4(uint4 a, uint4 b)
{
	uint4 c;
	c.x = a.x * FNV_PRIME ^ b.x;	
	c.y = a.y * FNV_PRIME ^ b.y;
	c.z = a.z * FNV_PRIME ^ b.z;
	c.w = a.w * FNV_PRIME ^ b.w;
	return c;
}

__device__ uint32_t fnv_reduce(uint4 v)
{
	return fnv(fnv(fnv(v.x, v.y), v.z), v.w);
}

__device__ hash64_t init_hash(hash32_t const* header, uint64_t nonce)
{
	hash64_t init;

	// sha3_512(header .. nonce)
	uint64_t state[25];

	copy(state, header->uint64s, 4);
	state[4] = nonce;
	state[5] = 0x0000000000000001;
	state[6] = 0;
	state[7] = 0;
	state[8] = 0x8000000000000000;
	for (uint32_t i = 9; i < 25; i++)
	{
		state[i] = 0;
	}
	
	keccak_f1600_block(state, 8);
	copy(init.uint64s, state, 8);
	return init;
}

__device__ uint32_t inner_loop(uint4 mix, uint32_t thread_id, uint32_t* share, hash128_t const* g_dag)
{
	// share init0
	if (thread_id == 0)
		*share = mix.x;

	uint32_t init0 = *share;
	
	uint32_t a = 0;

	do
	{
		
		bool update_share = thread_id == ((a >> 2) & (THREADS_PER_HASH-1));

		//#pragma unroll 4
		for (uint32_t i = 0; i < 4; i++)
		{

			if (update_share)
			{
				uint32_t m[4] = { mix.x, mix.y, mix.z, mix.w };
				*share = fnv(init0 ^ (a + i), m[i]) % d_dag_size;
			}
			__threadfence_block();

#if __CUDA_ARCH__ >= 350
			mix = fnv4(mix, __ldg(&g_dag[*share].uint4s[thread_id]));
#else
			mix = fnv4(mix, g_dag[*share].uint4s[thread_id]);
#endif
			
		}
		
	} while ((a += 4) != ACCESSES);
	
	return fnv_reduce(mix);
}

__device__ hash32_t final_hash(hash64_t const* init, hash32_t const* mix)
{
	uint64_t state[25];

	hash32_t hash;

	// keccak_256(keccak_512(header..nonce) .. mix);
	copy(state, init->uint64s, 8);
	copy(state + 8, mix->uint64s, 4);
	state[12] = 0x0000000000000001;
	for (uint32_t i = 13; i < 16; i++)
	{
		state[i] = 0;
	}
	state[16] = 0x8000000000000000;
	for (uint32_t i = 17; i < 25; i++)
	{
		state[i] = 0;
	}

	keccak_f1600_block(state,4);

	// copy out
	copy(hash.uint64s, state, 4);
	return hash;
}

typedef union
{
	hash64_t init;	
	hash32_t mix;
} compute_hash_share;



__device__ hash32_t compute_hash_shuffle(
	hash32_t const* g_header,
	hash128_t const* g_dag,
	uint64_t nonce
	)
{
	uint32_t s0,i0;
	uint32_t s1,i1;
	uint32_t s2,i2;
	uint32_t s3,i3;
	uint32_t s4,i4;
	uint32_t s5,i5;
	uint32_t s6,i6;
	uint32_t s7,i7;
	uint32_t s8,i8;
	uint32_t s9,i9;
	uint32_t s10,i10;
	uint32_t s11,i11;
	uint32_t s12,i12;
	uint32_t s13,i13;
	uint32_t s14,i14;
	uint32_t s15,i15;

	// sha3_512(header .. nonce)
	uint64_t state[25];

	copy(state, g_header->uint64s, 4);
	state[4] = nonce;
	state[5] = 0x0000000000000001;
	for (uint32_t i = 6; i < 25; i++)
	{
		state[i] = 0;
	}
	state[8] = 0x8000000000000000;
	keccak_f1600_block(state, 8);

	UNPACK64(i0, i1,	state[0]);
	UNPACK64(i2, i3,	state[1]);
	UNPACK64(i4, i5,	state[2]);
	UNPACK64(i6, i7,	state[3]);
	UNPACK64(i8, i9,	state[4]);
	UNPACK64(i10, i11,	state[5]);
	UNPACK64(i12, i13,	state[6]);
	UNPACK64(i14, i15,	state[7]);

	// Threads work together in this phase in groups of 8.
	uint32_t const thread_id = threadIdx.x & (THREADS_PER_HASH - 1);
	uint32_t const hash_id = threadIdx.x >> 3;

	int i = 0;
	int start_lane = hash_id << 3;

	do
	{
		s0 = __shfl(i0, start_lane + i);
		s1 = __shfl(i1, start_lane + i);
		s2 = __shfl(i2, start_lane + i);
		s3 = __shfl(i3, start_lane + i);
		s4 = __shfl(i4, start_lane + i);
		s5 = __shfl(i5, start_lane + i);
		s6 = __shfl(i6, start_lane + i);
		s7 = __shfl(i7, start_lane + i);
		s8 = __shfl(i8, start_lane + i);
		s9 = __shfl(i9, start_lane + i);
		s10 = __shfl(i10, start_lane + i);
		s11 = __shfl(i11, start_lane + i);
		s12 = __shfl(i12, start_lane + i);
		s13 = __shfl(i13, start_lane + i);
		s14 = __shfl(i14, start_lane + i);
		s15 = __shfl(i15, start_lane + i);

		uint4 mix;
		uint32_t t3 = thread_id & 3;
		if (t3 == 0) {
			mix = make_uint4(s0, s1, s2, s3);
		}
		else if (t3 == 1) {
			mix = make_uint4(s4, s5, s6, s7);
		}
		else if (t3 == 2) {
			mix = make_uint4(s8, s9, s10, s11);
		}
		else {
			mix = make_uint4(s12, s13, s14, s15);
		}
			
		s0 = (uint32_t)__shfl((int)(mix.x), start_lane);

		//uint32_t thread_mix = inner_loop_shuffle(thread_init, thread_id, &s0, g_dag, start_lane);

		uint32_t init0 = s0;
		uint32_t a = 0;

		do
		{
			int t = ((a >> 2) & (THREADS_PER_HASH - 1));

			//#pragma unroll 4
			for (uint32_t i = 0; i < 4; i++)
			{
				if (thread_id == t)
				{
					uint32_t m[4] = { mix.x, mix.y, mix.z, mix.w };
					s0 = fnv(init0 ^ (a + i), m[i]) % d_dag_size;
				}
				s0 = (uint32_t)__shfl((int)s0, start_lane + t);

#if __CUDA_ARCH__ >= 350
				mix = fnv4(mix, __ldg(&g_dag[s0].uint4s[thread_id]));
#else
				mix = fnv4(mix, g_dag[s0].uint4s[thread_id]);
#endif

			}

		} while ((a += 4) != ACCESSES);

		uint32_t thread_mix = fnv_reduce(mix);

		// update mix
		s0 = __shfl(thread_mix, start_lane + 0);
		s1 = __shfl(thread_mix, start_lane + 1);
		s2 = __shfl(thread_mix, start_lane + 2);
		s3 = __shfl(thread_mix, start_lane + 3);
		s4 = __shfl(thread_mix, start_lane + 4);
		s5 = __shfl(thread_mix, start_lane + 5);
		s6 = __shfl(thread_mix, start_lane + 6);
		s7 = __shfl(thread_mix, start_lane + 7);

		if (i == thread_id) {	
			//move mix into state:
			PACK64(state[8], s0, s1);
			PACK64(state[9], s2, s3);
			PACK64(state[10], s4, s5);
			PACK64(state[11], s6, s7);
		}
		
	} while (++i != THREADS_PER_HASH);

	hash32_t hash;

	// keccak_256(keccak_512(header..nonce) .. mix);
	state[12] = 0x0000000000000001;
	for (uint32_t i = 13; i < 25; i++)
	{
		state[i] = 0;
	}
	state[16] = 0x8000000000000000;
	keccak_f1600_block(state, 4);

	// copy out
	copy(hash.uint64s, state, 4);
	return hash;
}

__device__ hash32_t compute_hash(
	hash32_t const* g_header,
	hash128_t const* g_dag,
	uint64_t nonce
	)
{
	extern __shared__  compute_hash_share share[];

	// Compute one init hash per work item.
	hash64_t init = init_hash(g_header, nonce);

	// Threads work together in this phase in groups of 8.
	uint32_t const thread_id = threadIdx.x & (THREADS_PER_HASH-1);
	uint32_t const hash_id   = threadIdx.x >> 3;

	hash32_t mix;
	uint32_t i = 0;
	
	do
	{
		// share init with other threads
		if (i == thread_id)
			share[hash_id].init = init;
		
		uint4 thread_init = share[hash_id].init.uint4s[thread_id & 3];
		
		uint32_t thread_mix = inner_loop(thread_init, thread_id, share[hash_id].mix.uint32s, g_dag);

		share[hash_id].mix.uint32s[thread_id] = thread_mix;
		

		if (i == thread_id)
			mix = share[hash_id].mix;
		

	} while (++i != THREADS_PER_HASH );

	return final_hash(&init, &mix);
}

__global__ void 
__launch_bounds__(128, 7)
ethash_search(
	uint32_t* g_output,
	hash32_t const* g_header,
	hash128_t const* g_dag,
	uint64_t start_nonce,
	uint64_t target
	)
{

	uint32_t const gid = blockIdx.x * blockDim.x + threadIdx.x;	
	//hash32_t hash = compute_hash(g_header, g_dag, start_nonce + gid);
	hash32_t hash = compute_hash_shuffle(g_header, g_dag, start_nonce + gid);

	if (SWAP64(hash.uint64s[0]) < target)
	{
		atomicInc(g_output,d_max_outputs);
		g_output[g_output[0]] = gid;
	}
	
}

void run_ethash_hash(
	hash32_t* g_hashes,
	hash32_t const* g_header,
	hash128_t const* g_dag,
	uint64_t start_nonce
) 
{
}

void run_ethash_search(
	uint32_t blocks,
	uint32_t threads,
	hipStream_t stream,
	uint32_t* g_output,
	hash32_t const* g_header,
	hash128_t const* g_dag,
	uint64_t start_nonce,
	uint64_t target
)
{
//	ethash_search <<<blocks, threads, 0, stream >>>(g_output, g_header, g_dag, start_nonce, target);
	ethash_search <<<blocks, threads, (sizeof(compute_hash_share) * threads) / THREADS_PER_HASH, stream>>>(g_output, g_header, g_dag, start_nonce, target);
}

hipError_t set_constants(
	uint32_t * dag_size,
	uint32_t * max_outputs
	)
{
	hipError_t result;
	result = hipMemcpyToSymbol(HIP_SYMBOL(d_dag_size), dag_size, sizeof(uint32_t));
	result = hipMemcpyToSymbol(HIP_SYMBOL(d_max_outputs), max_outputs, sizeof(uint32_t));
	return result;
}
