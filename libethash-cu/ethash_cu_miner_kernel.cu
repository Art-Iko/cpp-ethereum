#include "hip/hip_runtime.h"
/*
* Genoil's CUDA mining kernel for Ethereum
* based on Tim Hughes' opencl kernel.
* thanks to trpuvot,djm34,sp,cbuchner for things i took from ccminer.
*/

#include "ethash_cu_miner_kernel.h"
#include "ethash_cu_miner_kernel_globals.h"
#include "rotl64.cuh"
#include ""
#include "hip/device_functions.h"
#include "hip/hip_vector_types.h"

#define ACCESSES 64
#define THREADS_PER_HASH (128 / 16)

#define FNV_PRIME	0x01000193

// Thanks for Lukas' code here
/*
#define SWAP64(n)					\
  (((n) << 56)						\
   | (((n) & 0xff00) << 40)			\
   | (((n) & 0xff0000) << 24)		\
   | (((n) & 0xff000000) << 8)		\
   | (((n) >> 8) & 0xff000000)		\
   | (((n) >> 24) & 0xff0000)		\
   | (((n) >> 40) & 0xff00)			\
   | ((n)  >> 56))
*/

#define SWAP64(v) \
  ((ROTL64L(v,  8) & 0x000000FF000000FF) | \
   (ROTL64L(v, 24) & 0x0000FF000000FF00) | \
   (ROTL64H(v, 40) & 0x00FF000000FF0000) | \
   (ROTL64H(v, 56) & 0xFF000000FF000000))

#define PACK64(result, lo, hi) asm("mov.b64 %0, {%1,%2};//pack64"  : "=l"(result) : "r"(lo), "r"(hi));
#define UNPACK64(lo, hi, input) asm("mov.b64 {%0, %1}, %2;//unpack64" : "=r"(lo),"=r"(hi) : "l"(input));


__device__ __constant__ uint64_t const keccak_round_constants[24] = {
	0x0000000000000001ULL, 0x0000000000008082ULL, 0x800000000000808AULL,
	0x8000000080008000ULL, 0x000000000000808BULL, 0x0000000080000001ULL,
	0x8000000080008081ULL, 0x8000000000008009ULL, 0x000000000000008AULL,
	0x0000000000000088ULL, 0x0000000080008009ULL, 0x000000008000000AULL,
	0x000000008000808BULL, 0x800000000000008BULL, 0x8000000000008089ULL,
	0x8000000000008003ULL, 0x8000000000008002ULL, 0x8000000000000080ULL,
	0x000000000000800AULL, 0x800000008000000AULL, 0x8000000080008081ULL,
	0x8000000000008080ULL, 0x0000000080000001ULL, 0x8000000080008008ULL
};

__device__ static void keccak_f1600_block(uint64_t* s, uint32_t out_size)//, uint32_t in_size, uint32_t out_size)
{
	uint64_t t[5], u, v;

	for (size_t i = 0; i < 24; i++) {
		/* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
		t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
		t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
		t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
		t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
		t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		/* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
		u = t[4] ^ ROTL64L(t[1], 1);
		s[0] ^= u; s[5] ^= u; s[10] ^= u; s[15] ^= u; s[20] ^= u;
		u = t[0] ^ ROTL64L(t[2], 1);
		s[1] ^= u; s[6] ^= u; s[11] ^= u; s[16] ^= u; s[21] ^= u;
		u = t[1] ^ ROTL64L(t[3], 1);
		s[2] ^= u; s[7] ^= u; s[12] ^= u; s[17] ^= u; s[22] ^= u;
		u = t[2] ^ ROTL64L(t[4], 1);
		s[3] ^= u; s[8] ^= u; s[13] ^= u; s[18] ^= u; s[23] ^= u;
		u = t[3] ^ ROTL64L(t[0], 1);
		s[4] ^= u; s[9] ^= u; s[14] ^= u; s[19] ^= u; s[24] ^= u;
		 
		/* rho pi: b[..] = rotl(a[..], ..) */
		u = s[1];

		s[1] = ROTL64H(s[6], 44);
		s[6] = ROTL64L(s[9], 20);
		s[9] = ROTL64H(s[22], 61);
		s[22] = ROTL64H(s[14], 39);
		s[14] = ROTL64L(s[20], 18);
		s[20] = ROTL64H(s[2], 62);
		s[2] = ROTL64H(s[12], 43);
		s[12] = ROTL64L(s[13], 25);
		s[13] = ROTL64L(s[19], 8);
		s[19] = ROTL64H(s[23], 56);
		s[23] = ROTL64H(s[15], 41);
		s[15] = ROTL64L(s[4], 27);
		s[4] = ROTL64L(s[24], 14);
		s[24] = ROTL64L(s[21], 2);
		s[21] = ROTL64H(s[8], 55);
		s[8] = ROTL64H(s[16], 45);
		s[16] = ROTL64H(s[5], 36);
		s[5] = ROTL64L(s[3], 28);
		s[3] = ROTL64L(s[18], 21);
		s[18] = ROTL64L(s[17], 15);
		s[17] = ROTL64L(s[11], 10);
		s[11] = ROTL64L(s[7], 6);
		s[7] = ROTL64L(s[10], 3);
		s[10] = ROTL64L(u, 1);

		// squeeze this in here
		/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
		u = s[0]; v = s[1]; s[0] ^= (~v) & s[2]; 
				
		/* iota: a[0,0] ^= round constant */
		s[0] ^= keccak_round_constants[i];
		if (i == 23 && out_size == 1) return;
		
		// continue chi
		s[1] ^= (~s[2]) & s[3]; s[2] ^= (~s[3]) & s[4]; s[3] ^= (~s[4]) & u; s[4] ^= (~u) & v;
		u = s[5]; v = s[6]; s[5] ^= (~v) & s[7]; s[6] ^= (~s[7]) & s[8]; s[7] ^= (~s[8]) & s[9]; 
		
		if (i == 23) return;
		s[8] ^= (~s[9]) & u; s[9] ^= (~u) & v;
		u = s[10]; v = s[11]; s[10] ^= (~v) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & u; s[14] ^= (~u) & v;
		u = s[15]; v = s[16]; s[15] ^= (~v) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & u; s[19] ^= (~u) & v;
		u = s[20]; v = s[21]; s[20] ^= (~v) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & u; s[24] ^= (~u) & v;
	}
}

#define copy(dst, src, count) for (uint32_t i = 0; i < count; i++) { (dst)[i] = (src)[i]; }

#define countof(x) (sizeof(x) / sizeof(x[0]))

#define fnv(x,y) ((x) * FNV_PRIME ^(y))

__device__ uint4 fnv4(uint4 a, uint4 b)
{
	uint4 c;
	c.x = a.x * FNV_PRIME ^ b.x;	
	c.y = a.y * FNV_PRIME ^ b.y;
	c.z = a.z * FNV_PRIME ^ b.z;
	c.w = a.w * FNV_PRIME ^ b.w;
	return c;
}

__device__ uint32_t fnv_reduce(uint4 v)
{
	return fnv(fnv(fnv(v.x, v.y), v.z), v.w);
}


__device__ hash64_t init_hash(hash32_t const* header, uint64_t nonce)
{
	hash64_t init;

	// sha3_512(header .. nonce)
	uint64_t state[25];

	copy(state, header->uint64s, 4);
	state[4] = nonce;
	state[5] = 0x0000000000000001;
	state[6] = 0;
	state[7] = 0;
	state[8] = 0x8000000000000000;
	for (uint32_t i = 9; i < 25; i++)
	{
		state[i] = 0;
	}
	
	keccak_f1600_block(state, 8);
	copy(init.uint64s, state, 8);
	return init;
}

__device__ uint32_t inner_loop(uint4 mix, uint32_t thread_id, uint32_t* share, hash128_t const* g_dag)
{
	// share init0
	if (thread_id == 0)
		*share = mix.x;

	uint32_t init0 = *share;
	
	uint32_t a = 0;

	do
	{
		
		bool update_share = thread_id == ((a >> 2) & (THREADS_PER_HASH-1));

		//#pragma unroll 4
		for (uint32_t i = 0; i < 4; i++)
		{

			if (update_share)
			{
				uint32_t m[4] = { mix.x, mix.y, mix.z, mix.w };
				*share = fnv(init0 ^ (a + i), m[i]) % d_dag_size;
			}
			__threadfence_block();

#if __CUDA_ARCH__ >= 350
			mix = fnv4(mix, __ldg(&g_dag[*share].uint4s[thread_id]));
#else
			mix = fnv4(mix, g_dag[*share].uint4s[thread_id]);
#endif
			
		}
		
	} while ((a += 4) != ACCESSES);
	
	return fnv_reduce(mix);
}

__device__ hash32_t final_hash(hash64_t const* init, hash32_t const* mix)
{
	uint64_t state[25];

	hash32_t hash;

	// keccak_256(keccak_512(header..nonce) .. mix);
	copy(state, init->uint64s, 8);
	copy(state + 8, mix->uint64s, 4);
	state[12] = 0x0000000000000001;
	for (uint32_t i = 13; i < 16; i++)
	{
		state[i] = 0;
	}
	state[16] = 0x8000000000000000;
	for (uint32_t i = 17; i < 25; i++)
	{
		state[i] = 0;
	}

	keccak_f1600_block(state,4);

	// copy out
	copy(hash.uint64s, state, 4);
	return hash;
}

typedef union
{
	hash64_t init;	
	hash32_t mix;
} compute_hash_share;

__device__ uint64_t compute_hash_shuffle(
	hash32_t const* g_header,
	hash128_t const* g_dag,
	uint64_t nonce
	)
{
	// sha3_512(header .. nonce)
	uint64_t state[25];
	
	copy(state, g_header->uint64s, 4);
	state[4] = nonce;
	state[5] = 0x0000000000000001;
	for (uint32_t i = 6; i < 25; i++)
	{
		state[i] = 0;
	}
	state[8] = 0x8000000000000000;
	keccak_f1600_block(state, 8);

	// Threads work together in this phase in groups of 8.
	uint64_t const thread_id = threadIdx.x & (THREADS_PER_HASH - 1);
	const int start_lane = (threadIdx.x >> 3) << 3;
	
	const uint32_t mix_idx = (thread_id & 3); 
	uint4 mix;

	uint32_t shuffle[16];
	uint32_t * init = (uint32_t *)state;

	for (int i = 0; i < THREADS_PER_HASH; i++)
	{

		// share init among threads
		for (int j = 0; j < 16; j++)
			shuffle[j] = __shfl(init[j], start_lane + i);
		
		// ugly but avoids local reads/writes
		if (mix_idx == 0) {
			mix = make_uint4(shuffle[0], shuffle[1], shuffle[2], shuffle[3]);			
		}
		else if (mix_idx == 1) {
			mix = make_uint4(shuffle[4], shuffle[5], shuffle[6], shuffle[7]);
		}
		else if (mix_idx == 2) {
			mix = make_uint4(shuffle[8], shuffle[9], shuffle[10], shuffle[11]);
		}
		else {
			mix = make_uint4(shuffle[12], shuffle[13], shuffle[14], shuffle[15]);
		}
		
		uint32_t init0 = __shfl(shuffle[0], start_lane);
		
		
		for (uint32_t a = 0; a < ACCESSES; a+=4)
		{
			int t = ((a >> 2) & (THREADS_PER_HASH - 1));

			for (uint32_t b = 0; b < 4; b++)
			{
				if (thread_id == t)
				{
					shuffle[0] = fnv(init0 ^ (a + b), ((uint32_t *)&mix)[b]) % d_dag_size;
				}
				
				shuffle[0] = __shfl(shuffle[0], start_lane + t);

				mix = fnv4(mix, g_dag[shuffle[0]].uint4s[thread_id]);			
			}
		} 

		uint32_t thread_mix = fnv_reduce(mix);

		// update mix accross threads

		for (int j = 0; j < 8; j++)
			shuffle[j] = __shfl(thread_mix, start_lane + j);

		if (i == thread_id) {	
			//move mix into state:
			PACK64(state[8],  shuffle[0], shuffle[1]);
			PACK64(state[9],  shuffle[2], shuffle[3]);
			PACK64(state[10], shuffle[4], shuffle[5]);
			PACK64(state[11], shuffle[6], shuffle[7]);
		}
		
	}

	// keccak_256(keccak_512(header..nonce) .. mix);
	state[12] = 0x0000000000000001;
	for (uint32_t i = 13; i < 25; i++)
	{
		state[i] = 0;
	}
	state[16] = 0x8000000000000000;
	keccak_f1600_block(state, 1);

	return state[0];
}


__device__ hash32_t compute_hash(
	hash32_t const* g_header,
	hash128_t const* g_dag,
	uint64_t nonce
	)
{
	extern __shared__  compute_hash_share share[];

	// Compute one init hash per work item.
	hash64_t init = init_hash(g_header, nonce);

	// Threads work together in this phase in groups of 8.
	uint32_t const thread_id = threadIdx.x & (THREADS_PER_HASH-1);
	uint32_t const hash_id   = threadIdx.x >> 3;

	hash32_t mix;
	uint32_t i = THREADS_PER_HASH;
	

	while (--i)
	{
		// share init with other threads
		if (i == thread_id)
			share[hash_id].init = init;
		
		uint4 thread_init = share[hash_id].init.uint4s[thread_id & 3];
		
		uint32_t thread_mix = inner_loop(thread_init, thread_id, share[hash_id].mix.uint32s, g_dag);

		share[hash_id].mix.uint32s[thread_id] = thread_mix;
		

		if (i == thread_id)
			mix = share[hash_id].mix;
	}

	return final_hash(&init, &mix);
}

__global__ void 
__launch_bounds__(128, 7)
ethash_search(
	uint32_t* g_output,
	hash32_t const* g_header,
	hash128_t const* g_dag,
	uint64_t start_nonce,
	uint64_t target
	)
{
	
	uint32_t const gid = blockIdx.x * blockDim.x + threadIdx.x;	
	/*
	hash32_t hash = compute_hash(g_header, g_dag, start_nonce + gid);	
	if (SWAP64(hash.uint64s[0]) < target)
	{
		atomicInc(g_output,d_max_outputs);
		g_output[g_output[0]] = gid;
	}
	*/
	
	uint64_t hash = compute_hash_shuffle(g_header, g_dag, start_nonce + gid);
	if (SWAP64(hash) < target)
	{
		atomicInc(g_output, d_max_outputs);
		g_output[g_output[0]] = gid;
	}
	
}

void run_ethash_hash(
	hash32_t* g_hashes,
	hash32_t const* g_header,
	hash128_t const* g_dag,
	uint64_t start_nonce
) 
{
}

void run_ethash_search(
	uint32_t blocks,
	uint32_t threads,
	hipStream_t stream,
	uint32_t* g_output,
	hash32_t const* g_header,
	hash128_t const* g_dag,
	uint64_t start_nonce,
	uint64_t target
)
{
	ethash_search <<<blocks, threads, 0, stream >>>(g_output, g_header, g_dag, start_nonce, target);
//	ethash_search <<<blocks, threads, (sizeof(compute_hash_share) * threads) / THREADS_PER_HASH, stream>>>(g_output, g_header, g_dag, start_nonce, target);
}

hipError_t set_constants(
	uint32_t * dag_size,
	uint32_t * max_outputs
	)
{
	hipError_t result;
	result = hipMemcpyToSymbol(HIP_SYMBOL(d_dag_size), dag_size, sizeof(uint32_t));
	result = hipMemcpyToSymbol(HIP_SYMBOL(d_max_outputs), max_outputs, sizeof(uint32_t));
	return result;
}
